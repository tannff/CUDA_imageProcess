#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <cmath>
#include <algorithm>

#define INF 1e20

/*CUDA�汾�ľ���任*/

//up to down, down to up
__device__ void likedt1dimvec(unsigned char* dim1data, unsigned char* dim1result, const int img_width, const int img_height)
{
	for (int i = 1; i != img_height; i++)
	{
		if (dim1data[i] > 0)
		{
			dim1result[i] = dim1data[i] + dim1result[i - 1];     //����Ԫ����ǰһ��Ԫ�ؽ����ۼ�
		}
	}
}

//left to right , right to left
__device__ void likedt1dimhor(unsigned char* dim1data, unsigned char* dim1result, const int img_width, const int img_height)
{
	for (int i = 1; i != img_width; i++)
	{
		if (dim1data[i] > 0)
		{
			dim1result[i] = dim1data[i] + dim1result[i - 1];
		}
	}
}

/*Ȼ���colpassimg���������һ�δ�����ɨ��*/
__global__ void left_to_rightpass(unsigned char* colpassimg, unsigned char* leftright, const int img_width, const int img_height)
{
	const int rows = 500;
	const int cols = 1216;
	
	//block�������̺߳��������һ�д�ȫ�ְ��˵������ڴ�
	__shared__ unsigned char rowdata[rows];
	int tid = threadIdx.x;
	
	while (tid < img_width)
	{
		int thid = tid + img_width * blockIdx.y;
		rowdata[tid] = colpassimg[thid];
		tid += blockDim.x;
	}
	__syncthreads();

	tid = threadIdx.x;
	//һ���߳� �Թ����ڴ�����ݽ���һά����任
	__shared__ unsigned char rowdataresult[rows];
	if (tid == 0)
	{
		likedt1dimhor(rowdata, rowdataresult, img_width, img_height);
	}
	__syncthreads();

	//block�������̺߳������������ڴ�ľ���任����ᵽȫ��ĳ��
	while (tid < img_width)
	{
		int thid = tid + img_width * blockIdx.y;
		leftright[thid] = rowdataresult[tid];
		tid += blockDim.x;
	}
}

/*Ȼ���colpassimg���������һ�δ��ҵ���ɨ��*/
__global__ void right_to_leftpass(unsigned char* colpassimg, unsigned char* rightleft, const int img_width, const int img_height)
{
	const int rows = 500;
	const int cols = 1216;

	//block�������̺߳��������һ�д�ȫ�ְ��˵������ڴ�
	__shared__ unsigned char rowdata[rows];
	int tid = threadIdx.x;
	while (tid < img_width)
	{
		int thid = tid + img_width * blockIdx.y;
		rowdata[img_width - 1 - tid] = colpassimg[thid];
		tid += blockDim.x;
	}
	__syncthreads();

	tid = threadIdx.x;
	//һ���߳� �Թ����ڴ�����ݽ���һά����任
	__shared__ unsigned char rowdataresult[rows];
	if (tid == 0)
	{
		likedt1dimhor(rowdata, rowdataresult, img_width, img_height);
	}
	__syncthreads();

	//block�������̺߳������������ڴ�ľ���任����ᵽȫ��ĳ��
	while (tid < img_width)
	{
		int thid = tid + img_width * blockIdx.y;
		rightleft[thid] = rowdataresult[img_width - 1 - tid];
		tid += blockDim.x;
	}
}


/*Ȼ���gpudtimg���������һ�δ��ϵ���ɨ��*/
__global__ void up_to_downscan(unsigned char* gpudtimg, unsigned char* updownpassimg, const int img_width, const int img_height)
{
	const int rows = 500;
	const int cols = 1216;
	int id = blockIdx.y * gridDim.x + blockIdx.x;
	int rowid = threadIdx.x;
	int globalid = id + img_width * rowid;

	//block�������̺߳��������һ�д�ȫ�ְ��˵������ڴ�
	__shared__ unsigned char coldata[cols];
	coldata[rowid] = gpudtimg[globalid];
	__syncthreads();


	//һ���߳� �Թ����ڴ�����ݽ���һά����任
	__shared__ unsigned char coldataresult[cols];
	if (rowid == 0)
	{
		likedt1dimvec(coldata, coldataresult, img_width, img_height);
	}
	__syncthreads();

	//block�������̺߳������������ڴ�ľ���任����ᵽȫ��ĳ��
	updownpassimg[globalid] = coldataresult[rowid];
}

/*Ȼ���gpudtimg���������һ�δ��µ���ɨ��*/
__global__ void down_to_upscan(unsigned char* gpudtimg, unsigned char* downuppassimg, const int img_width, const int img_height)
{
	const int rows = 500;
	const int cols = 1216;
	int id = blockIdx.y * gridDim.x + blockIdx.x;
	int rowid = threadIdx.x;
	int globalid = id + img_width * rowid;

	//block�������̺߳��������һ�д�ȫ�ְ��˵������ڴ�
	__shared__ unsigned char coldata[cols];
	coldata[img_height - 1 - rowid] = gpudtimg[globalid];
	__syncthreads();


	//һ���߳� �Թ����ڴ�����ݽ���һά����任
	__shared__ unsigned char coldataresult[cols];
	__syncthreads();
	if (rowid == 0)
	{
		likedt1dimvec(coldata, coldataresult, img_width, img_height);
	}
	__syncthreads();

	//block�������̺߳������������ڴ�ľ���任����ᵽȫ��ĳ��
	downuppassimg[globalid] = coldataresult[img_height - 1 - rowid];
}

/*����ͼ����ÿ�����ݿ����ţ��õ����ս��*/
__global__ void likedtresult(unsigned char* updown, unsigned char* downup, unsigned char* leftright, unsigned char* rightleft, unsigned char* dtresult, const int img_width, const int img_height)
{
	int rows = 500;
	int cols = 1216;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	int imgid = idy * rows + idx;

	unsigned char udvalue = updown[imgid];
	unsigned char minvalue = udvalue;
	unsigned char duvalue = downup[imgid];
	if (minvalue > duvalue)
	{
		minvalue = duvalue;
	}
	unsigned char lrvalue = leftright[imgid];
	if (minvalue > lrvalue)
	{
		minvalue = lrvalue;
	}
	unsigned char rlvalue = rightleft[imgid];
	if (minvalue > rlvalue)
	{
		minvalue = rlvalue;
	}

	dtresult[imgid] = minvalue;
}

extern hipError_t distancetransform(unsigned char* img_in, unsigned char* updown, unsigned char* downup, unsigned char* leftright, unsigned char* rightleft, unsigned char* dtresult, const int img_width, const int img_height) {

	dim3 block_dim(16, 16);   //�����߳̿�
	dim3 grid_dim = dim3((img_width + block_dim.x - 1) / block_dim.x,
		(img_height + block_dim.y - 1) / block_dim.y);

	left_to_rightpass << <grid_dim, block_dim, 0 >> > (img_in, leftright, img_width, img_height);
	right_to_leftpass << <grid_dim, block_dim, 0 >> > (img_in, rightleft, img_width, img_height);
	up_to_downscan << <grid_dim, block_dim, 0 >> > (img_in, updown, img_width, img_height);
	down_to_upscan << <grid_dim, block_dim, 0 >> > (img_in, downup, img_width, img_height);
	likedtresult << <grid_dim, block_dim, 0 >> > (updown, downup, leftright, rightleft, dtresult, img_width, img_height);
	return hipDeviceSynchronize();
}

__device__ int calculate_d4_distance(int2 pos_pre, int2 pos_target) {
	return abs(pos_pre.x - pos_target.x) + abs(pos_pre.y - pos_target.y);
}

__device__ int calculate_d8_distance(int2 pos_pre, int2 pos_target) {
	return max(abs(pos_pre.x - pos_target.x), abs(pos_pre.y - pos_target.y));
}

__global__ void kernel_distance_transform(unsigned char* img_in, unsigned char* img_out, const int img_width, const int img_height) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int idy = blockDim.y * blockIdx.y + threadIdx.y;

	if (idx >= img_width || idy >= img_height) {
		return;
	}

	int id = idy * img_width + idx;
	if (img_in[id] == 0) {
		img_out[id] = 0;
		return;
	}

	int min_distance = img_width * img_height;
	for (int i = 0; i < img_height; i++) {
		for (int j = 0; j < img_width; j++) {
			int cur_process_index = i * img_width + j;
			if (img_in[cur_process_index] == 0) {
				const int cur_distance = calculate_d4_distance(make_int2(idx, idy), make_int2(j, i));
				if (cur_distance == 1) {
					img_out[id] = 1;
					return;
				}
				min_distance = cur_distance < min_distance ? cur_distance : min_distance;
			}
		}
	}
	img_out[id] = min_distance >= 255 ? 255 : min_distance;
}



extern hipError_t distance_transform(unsigned char* img_in, unsigned char* img_out, const int img_width, const int img_height) {

	dim3 block_dim(16, 16);   //�����߳̿�
	dim3 grid_dim = dim3((img_width + block_dim.x - 1) / block_dim.x,
		(img_height + block_dim.y - 1) / block_dim.y);
	
	kernel_distance_transform<<<grid_dim, block_dim>>>(img_in, img_out, img_width, img_height);
	return hipDeviceSynchronize();
}
