#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <>
#include <opencv2/opencv.hpp>

#define STRONG_EDGE 0xFFFF
#define NON_EDGE 0x0

/* CUDA�汾�µı�Ե����㷨 */
//��˹�˲�
template<class T>
__device__ T clamp(T value, T min, T max) {
    T result;
    result = value < min ? min : value;
    result = value > max ? max : value;
    return result;
}

__global__ void kernel_gaussian_filter(unsigned char* img_in, unsigned char* img_gauss, int img_width, int img_height, int filterWidth, float* filter)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;    
    if (idx >= img_width || idy >= img_height) {
        return;
    }
    int local_index = idy * img_width + idx;
    float pixelcolor = 0.0f;

    for (int i = 0; i < filterWidth; i++)
    {
        for (int j = 0; j < filterWidth; j++)
        {
            // ������ͼ��߽�
            int clamp_x = __min(__max(idx + j - filterWidth / 2, 0), img_width - 1);
            int clamp_y = __min(__max(idy + i - filterWidth / 2, 0), img_height - 1);

            // �������
            float avg = filter[i * filterWidth + j];
            pixelcolor += avg * static_cast<float>(img_in[clamp_y * img_width + clamp_x]);
        }
    }
    // д����
    img_gauss[local_index] = clamp(pixelcolor, 0.f, 255.f);
}
extern hipError_t gaussian_filter(unsigned char* img_in, unsigned char* img_gauss, int img_width, int img_height, int filterWidth, float* filter) {

    dim3 block_dim(16, 16);   //�����߳̿�
    dim3 grid_dim = dim3((img_width + block_dim.x - 1) / block_dim.x,
        (img_height + block_dim.y - 1) / block_dim.y);
  
    kernel_gaussian_filter << <grid_dim, block_dim >> > (img_in, img_gauss, img_width, img_height, filterWidth, filter);
    return hipDeviceSynchronize();
}

//Sobel���Ӽ��������ݶ�
__global__ void kernel_sobel_intensity_gradient(unsigned char* img_in, unsigned char* img_sobel, int* Gx, int* Gy, int img_width, int img_height)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    if (idx >= img_width || idy >= img_height) {
        return;
    }
    int local_index = idy * img_width + idx;

    if (idx > 0 && idx < img_width - 1 && idy > 0 && idy < img_height - 1)
    { 
        Gx[local_index] = img_in[(idy - 1) * img_width + idx + 1] + 2.0 * img_in[idy * img_width + idx + 1] + img_in[(idy + 1) * img_width + idx + 1]       //idy * img_width + idxΪ��ǰ����
            - (img_in[(idy - 1) * img_width + idx - 1] + 2.0 * img_in[idy * img_width + idx - 1] + img_in[(idy + 1) * img_width + idx - 1]);
        Gy[local_index] = img_in[(idy - 1) * img_width + idx - 1] + 2.0 * img_in[(idy - 1) * img_width + idx] + img_in[(idy - 1) * img_width + idx + 1]
            - (img_in[(idy + 1) * img_width + idx - 1] + 2.0 * img_in[(idy + 1) * img_width + idx] + img_in[(idy + 1) * img_width + idx + 1]);
        img_sobel[local_index] = (abs(Gx[local_index]) + abs(Gy[local_index])) / 2.0;
    }
}
extern hipError_t sobel_intensity_gradient(unsigned char* img_in, unsigned char* img_sobel, int* Gx, int* Gy, int img_width, int img_height) {

    dim3 block_dim(16, 16);   //�����߳̿�
    dim3 grid_dim = dim3((img_width + block_dim.x - 1) / block_dim.x,
        (img_height + block_dim.y - 1) / block_dim.y);

    kernel_sobel_intensity_gradient << <grid_dim, block_dim >> > (img_in, img_sobel, Gx, Gy, img_width, img_height);
    return hipDeviceSynchronize();
}

//�Ǽ���ֵ����
__global__ void kernel_non_max(unsigned char* img_in, unsigned char* img_nms, int* totalGx, int* totalGy, int img_width, int img_height)
{
    const int SUPPRESSED = 0;

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    if (idx >= img_width || idy >= img_height) {
        return;
    }
    int local_index = idy * img_width + idx;
    float alpha;
    float mag1, mag2;
    int Gx = totalGx[local_index];
    int Gy = totalGy[local_index];

    if (idx == 0 || idx == (img_width - 1) || idy == 0 || idy == (img_height - 1)) {    //�ж��Ƿ���ͼ��߽�
        img_nms[local_index] = 0;
    }
    else  // �����ڱ߽���
    {         
      if (img_in == 0)     //���ݶȱ仯Ϊ0����Ҳ�����ڱ�Ե
          img_nms[local_index] = SUPPRESSED;
      else {
        if (Gx >= 0)  //Gx >= 0, Gy >= 0
          {
            if (Gy >= 0)
            {
                if ((Gx - Gy) >= 0)       // ����1�� SE-E
                {
                    alpha = (float)Gy / Gx;
                    mag1 = (1 - alpha) * img_in[idy * img_width + idx + 1] + alpha * img_in[(idy + 1) * img_width + idx + 1];
                    mag2 = (1 - alpha) * img_in[idy * img_width + idx - 1] + alpha * img_in[(idy - 1) * img_width + idx - 1];
                }
                else                      // ����2�� SE-S
                {
                    alpha = (float)Gx / Gy;
                    mag1 = (1 - alpha) * img_in[(idy + 1) * img_width + idx] + alpha * img_in[(idy + 1) * img_width + idx + 1];
                    mag2 = (1 - alpha) * img_in[(idy - 1) * img_width + idx] + alpha * img_in[(idy - 1) * img_width + idx - 1];
                }
            }
            else  //Gx >= 0, Gy < 0
            {
                if ((Gx + Gy) >= 0)       // ����8�� NE-E
                {
                    alpha = (float)-Gy / Gx;
                    mag1 = (1 - alpha) * img_in[idy * img_width + idx + 1] + alpha * img_in[(idy - 1) * img_width + idx + 1];
                    mag2 = (1 - alpha) * img_in[idy * img_width + idx - 1] + alpha * img_in[(idy + 1) * img_width + idx - 1];
                }
                else                      // ����7�� NE-N
                {
                    alpha = (float)Gx / -Gy;
                    mag1 = (1 - alpha) * img_in[(idy - 1) * img_width + idx] + alpha * img_in[(idy - 1) * img_width + idx + 1];
                    mag2 = (1 - alpha) * img_in[(idy + 1) * img_width + idx] + alpha * img_in[(idy - 1) * img_width + idx - 1];
                }
            }
        }
        else
        {
            if (Gy >= 0)  //Gx < 0, Gy >= 0
            {
                  if ((Gx + Gy) >= 0)    //  ����3�� SW-S
                  {
                      alpha = (float)-Gx / Gy;
                      mag1 = (1 - alpha) * img_in[(idy + 1) * img_width + idx] + alpha * img_in[(idy + 1) * img_width + idx - 1];
                      mag2 = (1 - alpha) * img_in[(idy - 1) * img_width + idx] + alpha * img_in[(idy - 1) * img_width + idx + 1];
                  }
                  else                   // ����4�� SW-W
                  {
                       alpha = (float)Gy / -Gx;
                       mag1 = (1 - alpha) * img_in[idy * img_width + idx - 1] + alpha * img_in[(idy + 1) * img_width + idx - 1];
                       mag2 = (1 - alpha) * img_in[idy * img_width + idx + 1] + alpha * img_in[(idy - 1) * img_width + idx + 1];
                  }
            }
            else    //Gx < 0, Gy < 0
            {
                  if ((-Gx + Gy) >= 0)   //  ����5�� NW-W
                  {
                       alpha = (float)Gy / Gx;
                       mag1 = (1 - alpha) * img_in[idy * img_width + idx - 1] + alpha * img_in[(idy - 1) * img_width + idx - 1];
                       mag2 = (1 - alpha) * img_in[idy * img_width + idx + 1] + alpha * img_in[(idy + 1) * img_width + idx + 1];
                  }
                  else                   //  ����6�� NW-N
                  {
                       alpha = (float)Gx / Gy;
                       mag1 = (1 - alpha) * img_in[(idy - 1) * img_width + idx] + alpha * img_in[(idy - 1) * img_width + idx - 1];
                       mag2 = (1 - alpha) * img_in[(idy + 1) * img_width + idx] + alpha * img_in[(idy + 1) * img_width + idx + 1];
                  }
            }
        }
        if ((img_in[local_index] < mag1) || (img_in[local_index] < mag2))
           img_nms[local_index] = SUPPRESSED;
        else
        {
            img_nms[local_index] = img_in[local_index]; 
        }
      } // END OF ELSE (mag != 0)
    } // END OF FOR(j)
} // END OF FOR(i)
extern hipError_t non_max(unsigned char* img_in, unsigned char* img_nms, int* Gx, int* Gy, int img_width, int img_height) {

    dim3 block_dim(16, 16);   //�����߳̿�
    dim3 grid_dim = dim3((img_width + block_dim.x - 1) / block_dim.x,
        (img_height + block_dim.y - 1) / block_dim.y);

    kernel_non_max << <grid_dim, block_dim >> > (img_in, img_nms, Gx, Gy, img_width, img_height);
    return hipDeviceSynchronize();
}
//��ֵ�ͺ���
__device__ void trace_immed_neighbors(unsigned char* img_in, unsigned char* img_trace, int idx, int idy, int t_low, int img_width, int img_height)
{
    unsigned n, s, e, w;   //��������
    unsigned nw, ne, sw, se;  //���ϣ����ϣ����£�����

    n = (idy - 1)* img_width + idx;
    nw = n - 1;
    ne = n + 1;
    s = (idy + 1) * img_width + idx;
    sw = s - 1;
    se = s + 1;
    w = idy * img_width + idx - 1;
    e = idy * img_width + idx + 1;

    if (img_in[nw] >= t_low) {
        img_trace[nw] = STRONG_EDGE;
    }
    if (img_in[n] >= t_low) {
        img_trace[n] = STRONG_EDGE;
    }
    if (img_in[ne] >= t_low) {
        img_trace[ne] = STRONG_EDGE;
    }
    if (img_in[w] >= t_low) {
        img_trace[w] = STRONG_EDGE;
    }
    if (img_in[e] >= t_low) {
        img_trace[e] = STRONG_EDGE;
    }
    if (img_in[sw] >= t_low) {
        img_trace[sw] = STRONG_EDGE;
    }
    if (img_in[s] >= t_low) {
        img_trace[s] = STRONG_EDGE;
    }
    if (img_in[se] >= t_low) {
        img_trace[se] = STRONG_EDGE;
    }
}
//�ͺ����ֵ
__global__ void kernel_hysteresis_high(unsigned char* img_in, unsigned char* img_trace, unsigned* strong_edge_mask, int t_high, int img_width, int img_height)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    if (idx >= img_width || idy >= img_height) {
        return;
    }
    int local_index = idy * img_width + idx;
    if (img_in[local_index] > t_high) {
        strong_edge_mask[local_index] = 1;
        img_trace[local_index] = STRONG_EDGE;
    }
    else {
        strong_edge_mask[local_index] = 0;
        img_trace[local_index] = NON_EDGE;
        }
    }
//�ͺ����ֵ
__global__ void kernel_hysteresis_low(unsigned char* img_in, unsigned char* img_trace, unsigned* strong_edge_mask, int t_low, int img_width, int img_height)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    if (idx >= img_width || idy >= img_height) {
        return;
    }
    int local_index = idy * img_width + idx;
    //�ų�����Խ��
    if ((idy > 0) && (idy < img_height - 1)     //�ų���β��
        && (idx > 0) && (idx < img_width - 1))  //�ų���β��
    {
        if (1 == strong_edge_mask[local_index]) { /* if this pixel was previously found to be a strong edge */
            trace_immed_neighbors(img_in, img_trace, idx, idy, t_low, img_width, img_height);
        }
    }
}
extern hipError_t hysteresis(unsigned char* img_in, unsigned char* img_high, unsigned char* img_trace, unsigned* strong_edge_mask, int t_high, int t_low, int img_width, int img_height) {

    dim3 block_dim(16, 16);   //�����߳̿�
    dim3 grid_dim = dim3((img_width + block_dim.x - 1) / block_dim.x,
        (img_height + block_dim.y - 1) / block_dim.y);

    kernel_hysteresis_high << <grid_dim, block_dim >> > (img_in, img_high, strong_edge_mask, t_high, img_width, img_height);
    kernel_hysteresis_low << <grid_dim, block_dim >> > (img_high, img_trace, strong_edge_mask, t_low, img_width, img_height);
    return hipDeviceSynchronize();
}